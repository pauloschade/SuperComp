#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <iostream>

int main() {
  thrust::host_vector<double> vcpu(2518);

  for(int i = 0; i < 2518; i++)
    std::cin >> vcpu[i];

  thrust::device_vector<double> vgpu(vcpu);

  double mean = thrust::reduce(vgpu.begin(), vgpu.end(), (double)0, thrust::plus<double>())/vgpu.size();
  std::cout << "Mean: " << mean << std::endl;

  //iterador que aponto inicialmente para a posicao end - 365 elementos
  thrust::device_vector<double>::iterator last_365 = vgpu.end() - 365;

  double mean_last_365 = thrust::reduce(last_365, vgpu.end(), (double)0, thrust::plus<double>())/365;
  std::cout << "Mean 365: " << mean_last_365 << std::endl;

  //iterator is a thrust pointer
  thrust::device_vector<double>::iterator max_el = thrust::max_element(vgpu.begin(), vgpu.end());
  thrust::device_vector<double>::iterator min_el = thrust::min_element(vgpu.begin(), vgpu.end());
  std::cout << "max element: " << *max_el << " | min element: " << *min_el <<  std::endl;

  return 0;
}