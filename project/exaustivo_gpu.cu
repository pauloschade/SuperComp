#include "hip/hip_runtime.h"
#include "helpers/helpers.h"
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <iostream>
#include <cassert>
#include <random>
#include <chrono>
#include <math.h>

using namespace std;

bool hasSlot(bool filled_slots[], movie curr) {
  if(curr.end == curr.start) return !filled_slots[curr.start];
  for(int i = curr.start; i < curr.end; i++) {
    if(filled_slots[i]) return false;
  }
  return true;
}

void fill_slot(bool filled_slots[], movie curr) {
  if(curr.end == curr.start) {
    filled_slots[curr.start] = true;
    return;
  }
  for(int i = curr.start; i < curr.end; i++) {
    filled_slots[i] = true;
  }
}

struct functor 
{
   int mov_count;
   int *movies;
   functor(int _mov_count, int *_movies) : mov_count(_mov_count), movies(_movies) {};
   __device__ 
   void operator() (const int selection_it)
   {
      bool slots[30];
      int added = 0;
      for(int i = 0; i < mov_count; i++)
      {
          if(added > 24) return -1;
          if(selection_it & (1 << i))
          {
            if(!hasSlot(movies[i], slots)) return -1;
            fill_slot(movies[i], slots);
            added++;
          }
      }
      return added;
   }
};

// bool check_limit(vector<movie> &selected, map<int, int> lim_cats, int n_cat) {
//   map<int, int> cats_count;
//   for(auto& mov: selected) {
//     if(lim_cats[mov.cat] == 0) return false;
//     lim_cats[mov.cat] --;
//   }
//   return true;
// }


chrono::steady_clock::time_point get_time() {
  return chrono::steady_clock::now();
}

//function to get chrono interval in seconds
double get_interval(chrono::steady_clock::time_point begin) {
  chrono::steady_clock::time_point end = get_time();
  return chrono::duration_cast<chrono::seconds>(end - begin).count();
}
//ref:
//https://stackoverflow.com/questions/43241174/javascript-generating-all-combinations-of-elements-in-a-single-array-in-pairs
void test_combinations(vector<movie> &movies, map<int, int> &lim_cats, int n_cat) {

  thrust::device_vector<movie> vgpu(movies.size());

  thrust::device_vector<int> mov_count(movies.size() ** 2);

  vgpu = movies;
  
  thrust::counting_iterator<int> comb(0);

  thrust::transform(comb.begin(), comb.end(), mov_count.begin(), functor(movies.size(), thrust::raw_pointer_cast(vgpu.data())));

  //get max element of mov_count
  thrust::device_vector<int>::iterator iter = thrust::max_element(mov_count.begin(), mov_count.end());

  int max = *iter;

  cout << "MAXXXXX" << max << endl;

}

int main(int argc, char *argv[]) {
  int n_mov, n_cat;
  // int id, n, w, weigth, val;

  map<int, bool> filled_slots;
  map<int, int> lim_cats;
  vector<movie> movies, selected;

  cin >> n_mov >> n_cat;
  
  movies.reserve(n_mov);

  read_cats_limit(lim_cats, n_cat);

  read_movies_data(movies, n_mov);

  test_combinations(movies, lim_cats, n_cat);

}