#include "hip/hip_runtime.h"
#include "helpers/helpers.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <iostream>
#include <cassert>
#include <random>
#include <chrono>
#include <math.h>

using namespace std;

struct functor 
{
   int mov_count;
   movie *movies;
   functor(int _mov_count, movie *_movies) : mov_count(_mov_count), movies(_movies) {};
   __device__
   int operator() (int selection_it)
   {
      bool slots[30];
      for(int h = 0; h < 30; h++) {
        slots[h] = false;
      } 

      int added = 0;
      for(int i = 0; i < mov_count; i++)
      {
          if(added > 24) return -1;
          if(selection_it & (1 << i))
          {
             if(movies[i].end == movies[i].start) return -1;
              for(int j = movies[i].start; j < movies[i].end; j++) {
                if(slots[j]) return -1;
              }
              for(int j = movies[i].start; j < movies[i].end; j++) {
                slots[j] = true;
              }
              // fill_slot(slots, movies[i]);
              added++;
          }
      }
      return added;
   }
};

// bool check_limit(vector<movie> &selected, map<int, int> lim_cats, int n_cat) {
//   map<int, int> cats_count;
//   for(auto& mov: selected) {
//     if(lim_cats[mov.cat] == 0) return false;
//     lim_cats[mov.cat] --;
//   }
//   return true;
// }


chrono::steady_clock::time_point get_time() {
  return chrono::steady_clock::now();
}

//function to get chrono interval in seconds
double get_interval(chrono::steady_clock::time_point begin) {
  chrono::steady_clock::time_point end = get_time();
  return chrono::duration_cast<chrono::seconds>(end - begin).count();
}
//ref:
//https://stackoverflow.com/questions/43241174/javascript-generating-all-combinations-of-elements-in-a-single-array-in-pairs
void test_combinations(vector<movie> &movies, map<int, int> &lim_cats, int n_cat) {

  thrust::device_vector<movie> vgpu(movies.size());

  thrust::device_vector<int> mov_count(pow(movies.size(), 2));

  vgpu = movies;
  
  thrust::counting_iterator<int> comb(0);

  thrust::transform(comb, comb +  pow(movies.size(), 2), mov_count.begin(), functor(movies.size(), thrust::raw_pointer_cast(vgpu.data())));

  //get max element of mov_count
  thrust::device_vector<int>::iterator iter = thrust::max_element(mov_count.begin(), mov_count.end());

  int max = *iter;

  cout << "MAXXXXX: " << max << endl;

}

int main(int argc, char *argv[]) {
  int n_mov, n_cat;
  // int id, n, w, weigth, val;

  map<int, bool> filled_slots;
  map<int, int> lim_cats;
  vector<movie> movies, selected;

  cin >> n_mov >> n_cat;
  
  movies.reserve(n_mov);

  read_cats_limit(lim_cats, n_cat);

  read_movies_data(movies, n_mov);

  test_combinations(movies, lim_cats, n_cat);

}