#include "hip/hip_runtime.h"
#include "helpers/helpers.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <iostream>
#include <cassert>
#include <random>
#include <chrono>
#include <math.h>

using namespace std;

struct mov_selection
{
  int selection_it;
  int mov_count;
  int n_cat;
  int *lim_cats;
  bool *slots;
  movie *movies;

  mov_selection(int _selection_it ,int _mov_count, movie *_movies, int _n_cat ,int *_lim_cats, bool *_slots) : selection_it(_selection_it), mov_count(_mov_count), movies(_movies), n_cat(_n_cat) ,lim_cats(_lim_cats), slots(_slots) {};

  __device__
  int operator() (int i)
  {
    if(selection_it & (1 << i))
    {
        if(movies[i].end == movies[i].start) {
          if(slots[movies[i].start]) return -1000;
          else slots[movies[i].start] = true;
        } 
        else {
          for(int j = movies[i].start; j < movies[i].end; j++) {
            if(slots[j]) return -1000;
            else slots[j] = true;
          }
        }
        //cout << movies[i].cat << endl;
        if(lim_cats[movies[i].cat] == 0) return -1;
        lim_cats[movies[i].cat] --;
        return 1;
    }
    return 0;
  }

  /* data */
};


struct functor 
{
   int mov_count;
   int n_cat;
   int *lim_cats;
   movie *movies;

  //  bool *slots;
  //  int *lim_cats_tmp;
  //  int *counting_it;

   functor(int _mov_count, movie *_movies, int _n_cat ,int *_lim_cats) : mov_count(_mov_count), movies(_movies), n_cat(_n_cat) ,lim_cats(_lim_cats) {
      // thrust::device_vector<bool> _slots(30);
      // thrust::fill(_slots.begin(), _slots.end(), false);
      // thrust::device_vector<int> _lim_cats_tmp(_n_cat+1);
      // thrust::copy(_lim_cats_tmp.begin(), _lim_cats_tmp.end(), _lim_cats);

      // thrust::counting_iterator<int> _counting_it(0);


      // slots = thrust::raw_pointer_cast(_slots.data());
      // lim_cats_tmp = thrust::raw_pointer_cast(_lim_cats_tmp.data());
   };
   __device__ __host__
   int operator() (int selection_it)
   {  
      // thrust::counting_iterator<int> comb(0);

      // int added = thrust::transform_reduce(comb, comb + mov_count, mov_selection(selection_it, mov_count, movies, n_cat, lim_cats_tmp, slots), 0, thrust::plus<int>());

      // return added;

      bool slots[30];
      for(int h = 0; h < 30; h++) {
        slots[h] = false;
      } 

      int lim_cats_cp[30];
      for(int h = 0; h < n_cat; h++) {
        lim_cats_cp[h] = lim_cats[h];
      }

      int added = 0;
      for(int i = 0; i < mov_count; i++)
      {
          if(added > 24) return -1;
          if(selection_it & (1 << i))
          {
              if(movies[i].end == movies[i].start) {
                if(slots[movies[i].start]) return -1;
                slots[movies[i].start] = true;
              } 
              else {
                for(int j = movies[i].start; j < movies[i].end; j++) {
                  if(slots[j]) return -1;
                  slots[j] = true;
                }
              }
              //cout << movies[i].cat << endl;
              if(lim_cats_cp[movies[i].cat] == 0) return -1;
              lim_cats_cp[movies[i].cat] --;
              added++;
          }
      }
      return added;
   }
};

// bool check_limit(vector<movie> &selected, map<int, int> lim_cats, int n_cat) {
//   map<int, int> cats_count;
//   for(auto& mov: selected) {
//     if(lim_cats[mov.cat] == 0) return false;
//     lim_cats[mov.cat] --;
//   }
//   return true;
// }


chrono::steady_clock::time_point get_time() {
  return chrono::steady_clock::now();
}

//function to get chrono interval in seconds
double get_interval(chrono::steady_clock::time_point begin) {
  chrono::steady_clock::time_point end = get_time();
  return chrono::duration_cast<chrono::seconds>(end - begin).count();
}
//ref:
//https://stackoverflow.com/questions/43241174/javascript-generating-all-combinations-of-elements-in-a-single-array-in-pairs
void test_combinations(vector<movie> &movies, vector<int> &lim_cats, int n_cat) {

  thrust::device_vector<int> cats_lim_gpu(lim_cats);
  //cats_lim_gpu = lim_cats;

  thrust::device_vector<movie> mov_gpu(movies);
  //mov_gpu = movies;

  thrust::device_vector<int> mov_count(pow(movies.size(), 2));
  
  thrust::counting_iterator<int> comb(0);

  thrust::transform(comb, comb + pow(movies.size(), 2), mov_count.begin(), functor(movies.size(), thrust::raw_pointer_cast(mov_gpu.data()), n_cat ,thrust::raw_pointer_cast(cats_lim_gpu.data())));

  //get max element of mov_count
  thrust::device_vector<int>::iterator iter = thrust::max_element(mov_count.begin(), mov_count.end());

  int max = *iter;

  cout << "MAX: " << max << endl;

}

int main(int argc, char *argv[]) {
  int n_mov, n_cat;
  // int id, n, w, weigth, val;
  vector<movie> movies, selected;
  vector<int> lim_cats;

  cin >> n_mov >> n_cat;
  
  movies.reserve(n_mov);

  read_cats_limit(lim_cats, n_cat);

  // for(int i = 0; i<=n_cat; i++) {
  //   cout << lim_cats[i] << ' ';
  // }

  cout << endl;

  read_movies_data(movies, n_mov);

  // for(int i=0; i < movies.size(); i++) cout << movies[i].cat << endl;

  test_combinations(movies, lim_cats, n_cat);

}