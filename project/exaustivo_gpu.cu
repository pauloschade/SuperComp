#include "hip/hip_runtime.h"
#include "helpers/helpers.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <iostream>
#include <cassert>
#include <random>
#include <chrono>
#include <math.h>

using namespace std;

struct mov_selection
{
  int selection_it;
  int mov_count;
  int n_cat;
  int *lim_cats;
  movie *movie;

  mov_selection(int _selection_it ,int _mov_count, movie *_movies, int _n_cat ,int *_lim_cats) : selection_it(_selection_it), mov_count(_mov_count), movies(_movies), n_cat(_n_cat) ,lim_cats(_lim_cats) {};

  __device__
  int operator() (int i)
  {
    if(selection_it & (1 << i))
    {
        if(movies[i].end == movies[i].start) {
          if(slots[movies[i].start]) return -1000;
          else slots[movies[i].start] = true;
        } 
        else {
          for(int j = movies[i].start; j < movies[i].end; j++) {
            if(slots[j]) return -1000;
            else slots[j] = true;
          }
        }
        //cout << movies[i].cat << endl;
        if(lim_cats_cp[movies[i].cat] == 0) return -1;
        lim_cats_cp[movies[i].cat] --;
        return 1;
    }
    return 0;
  }

  /* data */
};


struct functor 
{
   int mov_count;
   int n_cat;
   int *lim_cats;
   movie *movies;
   functor(int _mov_count, movie *_movies, int _n_cat ,int *_lim_cats) : mov_count(_mov_count), movies(_movies), n_cat(_n_cat) ,lim_cats(_lim_cats) {};
   __device__
   int operator() (int selection_it)
   {  
      thrust::device_vector<bool> slots(30);
      thrust::fill(slots.begin(), slots.end(), false);
      thrust::device_vector<int> lim_cats_tmp(n_cat+1);
      thrust::copy(lim_cats_tmp.begin(), lim_cats_tmp.end(), lim_cats);

      thrust::counting_iterator<int> comb(0);

      int added = thrust::transform_reduce(comb, comb + mov_count, mov_selection(selection_it, mov_count, movies, n_cat, thrust::raw_pointer_cast(lim_cats_tmp.data())), 0, thrust::plus<int>());

      return added;

      // int added = 0;
      // for(int i = 0; i < mov_count; i++)
      // {
      //     if(added > 24) return -1;
      //     if(selection_it & (1 << i))
      //     {
      //         if(movies[i].end == movies[i].start) {
      //           if(slots[movies[i].start]) return -1;
      //           else slots[movies[i].start] = true;
      //         } 
      //         else {
      //           for(int j = movies[i].start; j < movies[i].end; j++) {
      //             if(slots[j]) return -1;
      //             else slots[j] = true;
      //           }
      //         }
      //         //cout << movies[i].cat << endl;
      //         if(lim_cats_cp[movies[i].cat] == 0) return -1;
      //         lim_cats_cp[movies[i].cat] --;
      //         added++;
      //     }
      // }
      //return added;
   }
};

// bool check_limit(vector<movie> &selected, map<int, int> lim_cats, int n_cat) {
//   map<int, int> cats_count;
//   for(auto& mov: selected) {
//     if(lim_cats[mov.cat] == 0) return false;
//     lim_cats[mov.cat] --;
//   }
//   return true;
// }


chrono::steady_clock::time_point get_time() {
  return chrono::steady_clock::now();
}

//function to get chrono interval in seconds
double get_interval(chrono::steady_clock::time_point begin) {
  chrono::steady_clock::time_point end = get_time();
  return chrono::duration_cast<chrono::seconds>(end - begin).count();
}
//ref:
//https://stackoverflow.com/questions/43241174/javascript-generating-all-combinations-of-elements-in-a-single-array-in-pairs
void test_combinations(vector<movie> &movies, vector<int> &lim_cats, int n_cat) {

  thrust::device_vector<int> cats_lim_gpu(lim_cats);
  //cats_lim_gpu = lim_cats;

  thrust::device_vector<movie> mov_gpu(movies);
  //mov_gpu = movies;

  thrust::device_vector<int> mov_count(pow(movies.size(), 2));
  
  thrust::counting_iterator<int> comb(0);

  thrust::transform(comb, comb + pow(movies.size(), 2), mov_count.begin(), functor(movies.size(), thrust::raw_pointer_cast(mov_gpu.data()), n_cat ,thrust::raw_pointer_cast(cats_lim_gpu.data())));

  //get max element of mov_count
  thrust::device_vector<int>::iterator iter = thrust::max_element(mov_count.begin(), mov_count.end());

  int max = *iter;

  cout << "MAXXXXX: " << max << endl;

}

int main(int argc, char *argv[]) {
  int n_mov, n_cat;
  // int id, n, w, weigth, val;
  vector<movie> movies, selected;
  vector<int> lim_cats;

  cin >> n_mov >> n_cat;
  
  movies.reserve(n_mov);

  read_cats_limit(lim_cats, n_cat);

  for(int i = 0; i<=n_cat; i++) {
    cout << lim_cats[i] << ' ';
  }

  cout << endl;

  read_movies_data(movies, n_mov);

  for(int i=0; i < movies.size(); i++) cout << movies[i].cat << endl;

  test_combinations(movies, lim_cats, n_cat);

}