#include "hip/hip_runtime.h"
#include "helpers/helpers.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <iostream>
#include <cassert>
#include <random>
#include <chrono>
#include <math.h>

using namespace std;

struct functor 
{
   int mov_count;
   int n_cat;
   int *lim_cats;
   movie *movies;
   functor(int _mov_count, movie *_movies, int _n_cat, int *_lim_cats) : mov_count(_mov_count), movies(_movies), n_cat(_n_cat), lim_cats(lim_cats) {};
   __device__
   int operator() (int selection_it)
   {
      bool slots[30];
      for(int h = 0; h < 30; h++) {
        slots[h] = false;
      }

      int lim_cats_cp[n_cat];
      for(int h = 0; h < n_cat; h++) {
        lim_cats_cp[h+1] = lim_cats[h+1];
      }

      int added = 0;
      for(int i = 0; i < mov_count; i++)
      {
          if(added > 24) return -1;
          if(selection_it & (1 << i))
          {
              if(movies[i].end == movies[i].start) {
                if(slots[movies[i].start]) return -1;
                else filled_slots[movies[i].start] = true;
              } 
              else {
                for(int j = movies[i].start; j < movies[i].end; j++) {
                  if(slots[j]) return -1;
                  else filled_slots[j] = true;
                }
              }
              if(lim_cats_cp[movies[i].cat] == 0) return -1;
              lim_cats_cp[movies[i].cat] --;
              added++;
          }
      }
      return added;
   }
};

// bool check_limit(vector<movie> &selected, map<int, int> lim_cats, int n_cat) {
//   map<int, int> cats_count;
//   for(auto& mov: selected) {
//     if(lim_cats[mov.cat] == 0) return false;
//     lim_cats[mov.cat] --;
//   }
//   return true;
// }


chrono::steady_clock::time_point get_time() {
  return chrono::steady_clock::now();
}

//function to get chrono interval in seconds
double get_interval(chrono::steady_clock::time_point begin) {
  chrono::steady_clock::time_point end = get_time();
  return chrono::duration_cast<chrono::seconds>(end - begin).count();
}
//ref:
//https://stackoverflow.com/questions/43241174/javascript-generating-all-combinations-of-elements-in-a-single-array-in-pairs
void test_combinations(vector<movie> &movies, map<int, int> &lim_cats, int n_cat) {

  int lim_cats_cp[n_cat];
  for(int h = 0; h < n_cat; h++) {
    lim_cats_cp[h+1] = lim_cats[h+1];
  }

  thrust::device_vector<int> cats_lim_gpu(n_cat);
  cats_lim_gpu = lim_cats_cp;

  thrust::device_vector<movie> mov_gpu(movies.size());
  mov_gpu = movies;

  thrust::device_vector<int> mov_count(pow(movies.size(), 2));
  
  thrust::counting_iterator<int> comb(0);

  thrust::transform(comb, comb +  pow(movies.size(), 2), mov_count.begin(), functor(movies.size(), thrust::raw_pointer_cast(mov_gpu.data()), n_cat, thrust::raw_pointer_cast(cats_lim_gpu.data())));

  //get max element of mov_count
  thrust::device_vector<int>::iterator iter = thrust::max_element(mov_count.begin(), mov_count.end());

  int max = *iter;

  cout << "MAXXXXX: " << max << endl;

}

int main(int argc, char *argv[]) {
  int n_mov, n_cat;
  // int id, n, w, weigth, val;

  map<int, bool> filled_slots;
  map<int, int> lim_cats;
  vector<movie> movies, selected;

  cin >> n_mov >> n_cat;
  
  movies.reserve(n_mov);

  read_cats_limit(lim_cats, n_cat);

  read_movies_data(movies, n_mov);

  test_combinations(movies, lim_cats, n_cat);

}